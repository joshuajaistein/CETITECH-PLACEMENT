
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>


using std::cout;
using std::generate;
using std::vector;
using namespace std::chrono;


__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

int main() {

  int N = 4096;

  auto start = high_resolution_clock::now();


  size_t bytes = N * N * sizeof(int);

  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  int THREADS = 32;

  int BLOCKS = N / THREADS;

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);


  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
  cout << "\nMULTIPLICATION TIME : "<< duration.count() << " microseconds";


  return 0;
}
